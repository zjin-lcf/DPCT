
#include <inttypes.h>
#include <math.h>
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <iostream>
#include <assert.h>
#include <hip/hip_runtime.h>

#define AES_BLOCK_SIZE 16
#define THREADS_PER_BLOCK 256

#define cudaCHECK(code)                                                         \
  do {                                                                         \
    hipError_t cudaerr = code;                                                  \
    if (cudaerr != hipSuccess) {                                                \
      std::cerr << "ERROR on line " << __LINE__ << ": " << (unsigned)cudaerr    \
                << "\n";                                                       \
      abort();                                                                 \
    }                                                                          \
  } while (0)

#define F(x) (((x) << 1) ^ ((((x) >> 7) & 1) * 0x1b))
#define FD(x) (((x) >> 1) ^ (((x)&1) ? 0x8d : 0))

// S table
//__constant__ static const uint8_t sbox[256] = {
static const uint8_t sbox[256] = {
    0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b,
    0xfe, 0xd7, 0xab, 0x76, 0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0,
    0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0, 0xb7, 0xfd, 0x93, 0x26,
    0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
    0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2,
    0xeb, 0x27, 0xb2, 0x75, 0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0,
    0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84, 0x53, 0xd1, 0x00, 0xed,
    0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
    0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f,
    0x50, 0x3c, 0x9f, 0xa8, 0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5,
    0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2, 0xcd, 0x0c, 0x13, 0xec,
    0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
    0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14,
    0xde, 0x5e, 0x0b, 0xdb, 0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c,
    0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79, 0xe7, 0xc8, 0x37, 0x6d,
    0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
    0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f,
    0x4b, 0xbd, 0x8b, 0x8a, 0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e,
    0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e, 0xe1, 0xf8, 0x98, 0x11,
    0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
    0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f,
    0xb0, 0x54, 0xbb, 0x16};

// inv S table
//__constant__ static const uint8_t sboxinv[256] = {
static const uint8_t sboxinv[256] = {
    0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e,
    0x81, 0xf3, 0xd7, 0xfb, 0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87,
    0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb, 0x54, 0x7b, 0x94, 0x32,
    0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
    0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49,
    0x6d, 0x8b, 0xd1, 0x25, 0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16,
    0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92, 0x6c, 0x70, 0x48, 0x50,
    0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
    0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05,
    0xb8, 0xb3, 0x45, 0x06, 0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02,
    0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b, 0x3a, 0x91, 0x11, 0x41,
    0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
    0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8,
    0x1c, 0x75, 0xdf, 0x6e, 0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89,
    0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b, 0xfc, 0x56, 0x3e, 0x4b,
    0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
    0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59,
    0x27, 0x80, 0xec, 0x5f, 0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d,
    0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef, 0xa0, 0xe0, 0x3b, 0x4d,
    0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
    0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63,
    0x55, 0x21, 0x0c, 0x7d};

// x-time operation
__device__ uint8_t rj_xtime(uint8_t x) {
  return (x & 0x80) ? ((x << 1) ^ 0x1b) : (x << 1);
}

// subbyte operation
__device__ void aes_subBytes(uint8_t *buf, uint8_t *box) {
  uint8_t i, b;
  for (i = 0; i < 16; ++i) {
    b = buf[i];
    buf[i] = box[b];
  }
}

// inv subbyte operation
__device__ void aes_subBytes_inv(uint8_t *buf, uint8_t *boxinv) {
  uint8_t i, b;
  for (i = 0; i < 16; ++i) {
    b = buf[i];
    buf[i] = boxinv[b];
  }
}

// add round key operation
__device__ void aes_addRoundKey(uint8_t *buf, uint8_t *key) {
  uint8_t i = 16;
  while (i--) {
    buf[i] ^= key[i];
  }
}

// add round key at beginning
__device__ void aes_addRoundKey_cpy(uint8_t *buf, uint8_t *key, uint8_t *cpk) {
  uint8_t i = 16;
  while (i--) {
    buf[i] ^= (cpk[i] = key[i]);
    cpk[16 + i] = key[16 + i];
  }
}

__device__ void memcpy_u8(uint8_t *dst, uint8_t *src, uint32_t size) {
  for (uint32_t i = 0; i < size; i++)
    dst[i] = src[i];
}

// shift row operation
__device__ void aes_shiftRows(uint8_t *buf) {
  uint8_t i, j;
  i = buf[1];
  buf[1] = buf[5];
  buf[5] = buf[9];
  buf[9] = buf[13];
  buf[13] = i;
  i = buf[10];
  buf[10] = buf[2];
  buf[2] = i;
  j = buf[3];
  buf[3] = buf[15];
  buf[15] = buf[11];
  buf[11] = buf[7];
  buf[7] = j;
  j = buf[14];
  buf[14] = buf[6];
  buf[6] = j;
}

// inv shift row operation
__device__ void aes_shiftRows_inv(uint8_t *buf) {
  uint8_t i, j;
  i = buf[1];
  buf[1] = buf[13];
  buf[13] = buf[9];
  buf[9] = buf[5];
  buf[5] = i;
  i = buf[2];
  buf[2] = buf[10];
  buf[10] = i;
  j = buf[3];
  buf[3] = buf[7];
  buf[7] = buf[11];
  buf[11] = buf[15];
  buf[15] = j;
  j = buf[6];
  buf[6] = buf[14];
  buf[14] = j;
}

// mix column operation
__device__ void aes_mixColumns(uint8_t *buf) {
  uint8_t i, a, b, c, d, e;
  for (i = 0; i < 16; i += 4) {
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    buf[i] ^= e ^ rj_xtime(a ^ b);
    buf[i + 1] ^= e ^ rj_xtime(b ^ c);
    buf[i + 2] ^= e ^ rj_xtime(c ^ d);
    buf[i + 3] ^= e ^ rj_xtime(d ^ a);
  }
}

// inv mix column operation
__device__ void aes_mixColumns_inv(uint8_t *buf) {
  uint8_t i, a, b, c, d, e, x, y, z;
  for (i = 0; i < 16; i += 4) {
    a = buf[i];
    b = buf[i + 1];
    c = buf[i + 2];
    d = buf[i + 3];
    e = a ^ b ^ c ^ d;
    z = rj_xtime(e);
    x = e ^ rj_xtime(rj_xtime(z ^ a ^ c));
    y = e ^ rj_xtime(rj_xtime(z ^ b ^ d));
    buf[i] ^= x ^ rj_xtime(a ^ b);
    buf[i + 1] ^= y ^ rj_xtime(b ^ c);
    buf[i + 2] ^= x ^ rj_xtime(c ^ d);
    buf[i + 3] ^= y ^ rj_xtime(d ^ a);
  }
}

// add expand key operation
__device__ __host__ void aes_expandEncKey(uint8_t *k, uint8_t *rc,
                                          const uint8_t *sb) {
  uint8_t i;

  k[0] ^= sb[k[29]] ^ (*rc);
  k[1] ^= sb[k[30]];
  k[2] ^= sb[k[31]];
  k[3] ^= sb[k[28]];
  *rc = F(*rc);

  for (i = 4; i < 16; i += 4) {
    k[i] ^= k[i - 4];
    k[i + 1] ^= k[i - 3];
    k[i + 2] ^= k[i - 2];
    k[i + 3] ^= k[i - 1];
  }

  k[16] ^= sb[k[12]];
  k[17] ^= sb[k[13]];
  k[18] ^= sb[k[14]];
  k[19] ^= sb[k[15]];

  for (i = 20; i < 32; i += 4) {
    k[i] ^= k[i - 4];
    k[i + 1] ^= k[i - 3];
    k[i + 2] ^= k[i - 2];
    k[i + 3] ^= k[i - 1];
  }
}

// inv add expand key operation
__device__ void aes_expandDecKey(uint8_t *k, uint8_t *rc, uint8_t *box) {
  uint8_t i;

  for (i = 28; i > 16; i -= 4) {
    k[i + 0] ^= k[i - 4];
    k[i + 1] ^= k[i - 3];
    k[i + 2] ^= k[i - 2];
    k[i + 3] ^= k[i - 1];
  }

  k[16] ^= box[k[12]];
  k[17] ^= box[k[13]];
  k[18] ^= box[k[14]];
  k[19] ^= box[k[15]];

  for (i = 12; i > 0; i -= 4) {
    k[i + 0] ^= k[i - 4];
    k[i + 1] ^= k[i - 3];
    k[i + 2] ^= k[i - 2];
    k[i + 3] ^= k[i - 1];
  }

  *rc = FD(*rc);
  k[0] ^= box[k[29]] ^ (*rc);
  k[1] ^= box[k[30]];
  k[2] ^= box[k[31]];
  k[3] ^= box[k[28]];
}

// key initition
void aes256_init(uint8_t *k, uint8_t *ctx_key, uint8_t *ctx_enckey,
                 uint8_t *ctx_deckey) {
  uint8_t rcon = 1;
  uint8_t i;

  for (i = 0; i < 32; i++) {
    ctx_enckey[i] = ctx_deckey[i] = k[i];
  }
  for (i = 8; --i;) {
    aes_expandEncKey(ctx_deckey, &rcon, sbox);
  }
}

// aes encrypt algorithm one thread/one block with AES_BLOCK_SIZE
__global__ void aes256_encrypt_ecb(uint8_t *buf_d, unsigned long numbytes,
                                   uint8_t *ctx_enckey_d, uint8_t *ctx_key_d,
                                   uint8_t *sbox_d) {
  uint8_t i, rcon;
  uint8_t buf_t[AES_BLOCK_SIZE]; // thread buffer
  // printf("Thread %d\n", threadIdx.x);
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) +
                         (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) {
    return;
  }

  memcpy_u8(buf_t, &buf_d[offset], AES_BLOCK_SIZE);
  aes_addRoundKey_cpy(buf_t, ctx_enckey_d, ctx_key_d);
  for (i = 1, rcon = 1; i < 14; ++i) {
    aes_subBytes(buf_t, sbox_d);
    aes_shiftRows(buf_t);
    aes_mixColumns(buf_t);
    if (i & 1) {
      aes_addRoundKey(buf_t, &ctx_key_d[16]);
    } else {
      aes_expandEncKey(ctx_key_d, &rcon, sbox_d),
          aes_addRoundKey(buf_t, ctx_key_d);
    }
  }
  aes_subBytes(buf_t, sbox_d);
  aes_shiftRows(buf_t);
  aes_expandEncKey(ctx_key_d, &rcon, sbox_d);
  aes_addRoundKey(buf_t, ctx_key_d);
  /* copy thread buffer back into global memory */
  memcpy_u8(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
}

// aes decrypt algorithm
__global__ void aes256_decrypt_ecb(uint8_t *buf_d, unsigned long numbytes,
                                   uint8_t *ctx_deckey_d, uint8_t *ctx_key_d,
                                   uint8_t *sbox_d, uint8_t *sboxinv_d) {
  uint8_t i, rcon;
  uint8_t buf_t[AES_BLOCK_SIZE];
  unsigned long offset = (blockIdx.x * THREADS_PER_BLOCK * AES_BLOCK_SIZE) +
                         (threadIdx.x * AES_BLOCK_SIZE);
  if (offset >= numbytes) {
    return;
  }
  memcpy_u8(buf_t, &buf_d[offset], AES_BLOCK_SIZE);
  aes_addRoundKey_cpy(buf_t, ctx_deckey_d, ctx_key_d);
  aes_shiftRows_inv(buf_t);
  aes_subBytes_inv(buf_t, sboxinv_d);
  for (i = 14, rcon = 0x80; --i;) {
    if ((i & 1)) {
      aes_expandDecKey(ctx_key_d, &rcon, sbox_d);
      aes_addRoundKey(buf_t, &ctx_key_d[16]);
    } else {
      aes_addRoundKey(buf_t, ctx_key_d);
    }
    aes_mixColumns_inv(buf_t);
    aes_shiftRows_inv(buf_t);
    aes_subBytes_inv(buf_t, sboxinv_d);
  }
  aes_addRoundKey(buf_t, ctx_key_d);
  /* copy thread back into global memory */
  memcpy_u8(&buf_d[offset], buf_t, AES_BLOCK_SIZE);
  __syncthreads();
}

// aes encrypt demo
float encryptdemo(uint8_t *buf, unsigned long numbytes, bool measure) {
  uint8_t key[32];
  uint8_t ctx_key[32];
  uint8_t ctx_enckey[32];
  uint8_t ctx_deckey[32];
  uint8_t sboxy[256];
  /*
    uint8_t *key = (uint8_t *)malloc(32);
    uint8_t *ctx_key = (uint8_t *)malloc(32);
    uint8_t *ctx_enckey = (uint8_t *)malloc(32);
    uint8_t *ctx_deckey = (uint8_t *)malloc(32);
    uint8_t *sboxy = (uint8_t *)malloc(256);
  */
  for (unsigned i = 0; i < 32; i++)
    key[i] = i;
  memcpy(sboxy, sbox, 256);

  uint8_t *buf_d = NULL;
  uint8_t *ctx_key_d = NULL;
  uint8_t *ctx_enckey_d = NULL;
  uint8_t *sbox_d = NULL;
  hipEvent_t start, stop;
  float retval = 0.0f;

  printf("\nBeginning encryption\n");
  aes256_init(key, ctx_key, ctx_enckey, ctx_deckey);

  cudaCHECK(hipMalloc((void **)&buf_d, numbytes));
  assert(buf_d);
  cudaCHECK(hipMalloc((void **)&ctx_enckey_d, sizeof(ctx_enckey)));
  assert(ctx_enckey_d);
  cudaCHECK(hipMalloc((void **)&ctx_key_d, sizeof(ctx_key)));
  assert(ctx_key_d);
  cudaCHECK(hipMalloc((void **)&sbox_d, sizeof(sboxy)));
  assert(sbox_d);

  cudaCHECK(hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice));
  cudaCHECK(hipMemcpy(sbox_d, sboxy, sizeof(sboxy), hipMemcpyHostToDevice));
  cudaCHECK(hipMemcpy(ctx_enckey_d, ctx_enckey, sizeof(ctx_enckey),
                     hipMemcpyHostToDevice));
  cudaCHECK(
      hipMemcpy(ctx_key_d, ctx_key, sizeof(ctx_key), hipMemcpyHostToDevice));

  dim3 dimBlock(
      ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  if (measure) {
    cudaCHECK(hipEventCreate(&start));
    cudaCHECK(hipEventRecord(start));
  }
  // printf("Creating %d threads over %d blocks\n", dimBlock.x*dimGrid.x,
  // dimBlock.x);
  aes256_encrypt_ecb<<<dim3(dimBlock), dim3(dimGrid)>>>(
                     buf_d, numbytes, ctx_enckey_d, ctx_key_d, sbox_d);
  cudaCHECK(hipGetLastError());
  cudaCHECK(hipDeviceSynchronize());
  if (measure) {
    cudaCHECK(hipEventCreate(&stop));
    cudaCHECK(hipEventRecord(stop));
  }

  cudaCHECK(hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost));
  // print(buf);
  cudaCHECK(hipMemcpy(ctx_enckey, ctx_enckey_d, sizeof(ctx_enckey),
                     hipMemcpyDeviceToHost));
  cudaCHECK(
      hipMemcpy(ctx_key, ctx_key_d, sizeof(ctx_key), hipMemcpyDeviceToHost));

  cudaCHECK(hipDeviceSynchronize());
  if (measure) {
    cudaCHECK(hipEventElapsedTime(&retval, start, stop));
  }

  cudaCHECK(hipFree(buf_d));
  cudaCHECK(hipFree(ctx_key_d));
  cudaCHECK(hipFree(ctx_enckey_d));
  cudaCHECK(hipFree(sbox_d));
  if (measure) {
    cudaCHECK(hipEventDestroy(start));
    cudaCHECK(hipEventDestroy(stop));
  }
  return retval;
}

// aes decrypt demo
float decryptdemo(uint8_t *buf, unsigned long numbytes, bool measure) {
  uint8_t key[32];
  uint8_t ctx_key[32];
  uint8_t ctx_enckey[32];
  uint8_t ctx_deckey[32];
  uint8_t sboxy[256];
  uint8_t sboxinvy[256];
  /*
    uint8_t *key = (uint8_t *)malloc(32);
    uint8_t *ctx_key = (uint8_t *)malloc(32);
    uint8_t *ctx_enckey = (uint8_t *)malloc(32);
    uint8_t *ctx_deckey = (uint8_t *)malloc(32);
    uint8_t *sboxy = (uint8_t *)malloc(256);
    uint8_t *sboxinvy = (uint8_t *)malloc(256);
  */
  for (unsigned i = 0; i < 32; i++)
    key[i] = i;
  memcpy(sboxy, sbox, 256);
  memcpy(sboxinvy, sboxinv, 256);

  uint8_t *buf_d = NULL;
  uint8_t *ctx_key_d = NULL, *ctx_deckey_d = NULL;
  uint8_t *sbox_d = NULL;
  uint8_t *sboxinv_d = NULL;
  hipEvent_t start, stop;
  float retval = 0.0f;

  // cudaMemcpyToSymbol(cuda_SYMBOL(sboxinv), sboxinv, sizeof(uint8_t)*256);

  printf("\nBeginning decryption\n");
  aes256_init(key, ctx_key, ctx_enckey, ctx_deckey);

  cudaCHECK(hipMalloc((void **)&buf_d, numbytes));
  cudaCHECK(hipMalloc((void **)&ctx_deckey_d, sizeof(ctx_deckey)));
  cudaCHECK(hipMalloc((void **)&ctx_key_d, sizeof(ctx_key)));
  cudaCHECK(hipMalloc((void **)&sbox_d, sizeof(sboxy)));
  cudaCHECK(hipMalloc((void **)&sboxinv_d, sizeof(sboxinv)));

  cudaCHECK(hipMemcpy(buf_d, buf, numbytes, hipMemcpyHostToDevice));
  cudaCHECK(hipMemcpy(sbox_d, sboxy, sizeof(sboxy), hipMemcpyHostToDevice));
  cudaCHECK(
      hipMemcpy(sboxinv_d, sboxinvy, sizeof(sboxinv), hipMemcpyHostToDevice));
  cudaCHECK(hipMemcpy(ctx_deckey_d, ctx_deckey, sizeof(ctx_deckey),
                     hipMemcpyHostToDevice));
  cudaCHECK(
      hipMemcpy(ctx_key_d, ctx_key, sizeof(ctx_key), hipMemcpyHostToDevice));

  if (measure) {
    cudaCHECK(hipEventCreate(&start));
    cudaCHECK(hipEventRecord(start));
  }
  dim3 dimBlock(
      ceil((double)numbytes / (double)(THREADS_PER_BLOCK * AES_BLOCK_SIZE)));
  dim3 dimGrid(THREADS_PER_BLOCK);
  aes256_decrypt_ecb<<<dim3(dimBlock), dim3(dimGrid)>>>(
                     buf_d, numbytes, ctx_deckey_d, ctx_key_d, sbox_d,
                     sboxinv_d);
  cudaCHECK(hipGetLastError());
  cudaCHECK(hipDeviceSynchronize());
  if (measure) {
    cudaCHECK(hipEventCreate(&stop));
    cudaCHECK(hipEventRecord(stop));
  }

  cudaCHECK(hipMemcpy(buf, buf_d, numbytes, hipMemcpyDeviceToHost));
  cudaCHECK(hipMemcpy(ctx_deckey, ctx_deckey_d, sizeof(ctx_deckey),
                     hipMemcpyDeviceToHost));
  cudaCHECK(
      hipMemcpy(ctx_key, ctx_key_d, sizeof(ctx_key), hipMemcpyDeviceToHost));

  cudaCHECK(hipDeviceSynchronize());
  if (measure) {
    cudaCHECK(hipEventElapsedTime(&retval, start, stop));
  }

  cudaCHECK(hipFree(buf_d));
  cudaCHECK(hipFree(ctx_key_d));
  cudaCHECK(hipFree(ctx_deckey_d));
  cudaCHECK(hipFree(sbox_d));
  cudaCHECK(hipFree(sboxinv_d));
  if (measure) {
    cudaCHECK(hipEventDestroy(start));
    cudaCHECK(hipEventDestroy(stop));
  }
  return retval;
}


int main() {

  // open file
  FILE *file;
  uint8_t *buf;
  size_t numbytes;
  const char *fname;
  int mili_sec, i;
  size_t padded_size;

  int deviceCount = 0;
  cudaCHECK(hipGetDeviceCount(&deviceCount));

  if (deviceCount == 0) {
    printf("There are no available device(s) that support CUDA\n");
    exit(EXIT_FAILURE);
  }

  // handle txt file
  fname = "input.txt";
  file = fopen(fname, "r");
  if (file == NULL) {
    printf("input file %s doesn't exist\n", fname);
    exit(1);
  }
  printf("Opened file %s\n", fname);
  fseek(file, 0L, SEEK_END);
  numbytes = ftell(file);
  printf("Size is %lu\n", numbytes);
  assert(numbytes > 0);

  // calculate the padding
  padded_size = numbytes;
  if ((numbytes & (AES_BLOCK_SIZE - 1)) > 0)
    padded_size = (numbytes | (AES_BLOCK_SIZE - 1)) + 1;
  printf("Padding file for a new size of %lu\n", padded_size);

  // copy file into memory
  fseek(file, 0L, SEEK_SET);
  buf = (uint8_t *)calloc(padded_size, sizeof(uint8_t));
  if (buf == NULL)
    exit(1);
  if (fread(buf, 1, numbytes, file) != numbytes) {
    printf("Unable to read all bytes from file %s\n", fname);
    exit(EXIT_FAILURE);
  }
  fclose(file);

  // encryption
  encryptdemo(buf, padded_size, false);
  encryptdemo(buf, padded_size, false);
  float enctime = encryptdemo(buf, padded_size, true);

  // write into file
  /*
    file = fopen("gpu_cipher.txt", "w");
    fwrite(buf, 1, padded_size, file);
    fclose(file);
  */
  // decryption
  decryptdemo(buf, padded_size, false);
  decryptdemo(buf, padded_size, false);
  float dectime = decryptdemo(buf, padded_size, true);

  // write into file
  /*
    file = fopen("gpu_output.txt", "w");
    fwrite(buf, 1, numbytes, file);
    fclose(file);
  */

  printf("Encryption time: %f ms\n", enctime);
  printf("GPU encryption throughput: %f KB/second\n",
         (float)padded_size / enctime);

  printf("Decryption time: %f ms\n", dectime);
  printf("GPU decryption throughput: %f KB/second\n",
         (float)padded_size / dectime);

  free(buf);
  return EXIT_SUCCESS;
}
