/******************************************************************************
LICENSE

Copyright (c) 2015 Codeplay Software Ltd.
Copyright (c) 2006-2008 Kevin Beason (kevin.beason@gmail.com)

Permission is hereby granted, free of charge, to any person obtaining
a copy of this software and associated documentation files (the
"Software"), to deal in the Software without restriction, including
without limitation the rights to use, copy, modify, merge, publish,
distribute, sublicense, and/or sell copies of the Software, and to
permit persons to whom the Software is furnished to do so, subject to
the following conditions:

The above copyright notice and this permission notice shall be included
in all copies or substantial portions of the Software.

THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND,
EXPRESS OR IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF
MERCHANTABILITY, FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.
IN NO EVENT SHALL THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY
CLAIM, DAMAGES OR OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT,
TORT OR OTHERWISE, ARISING FROM, OUT OF OR IN CONNECTION WITH THE
SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
 *
 * Modified version for SYCL of Kevin Beason smallpt
 * http://www.kevinbeason.com/smallpt/
 *
 *
 *
 * Modified version using CUDA
 *   Add the original License in the source file
 *   The number of samples is set at 40
 *   spheres_glob is not declared in the global space
 *
 * Zheming Jin
 *
 ******************************************************************************/

#include <cstdio>
#include <cstdlib>
#include <cmath>
#include <hip/hip_runtime.h>


class RNG {
	public:
		unsigned int x;
		const uint32_t fmask = (1 << 23) - 1;

	        __device__
		RNG(const unsigned int seed) { x = seed; }

	        __device__
		uint32_t next() {
			x ^= x >> 6;
			x ^= x << 17;
			x ^= x >> 9;
			return uint32_t(x);
		}
	        __device__
		float operator()(void) {
			union {
				float f;
				uint32_t i;
			} u;
			u.i = (next() & fmask) | 0x3f800000;
			return u.f - 1.f;
		}
};

struct Vec {     // Usage: time ./smallpt 5000 && xv image.ppm
	float x, y, z; // position, also color (r,g,b)
	__host__ __device__
	Vec(float x_ = 0, float y_ = 0, float z_ = 0) : x(x_), y(y_), z(z_) {}
	__device__
	Vec operator+(const Vec &b) const { return Vec(x + b.x, y + b.y, z + b.z); }
	__device__
	Vec operator-(const Vec &b) const { return Vec(x - b.x, y - b.y, z - b.z); }
	__host__ __device__
	Vec operator*(float b) const { return Vec(x * b, y * b, z * b); }
	__device__
	Vec mult(const Vec &b) const { return Vec(x * b.x, y * b.y, z * b.z); }
	__device__
	Vec &norm() {
		return *this = *this * (1 / sqrt(x * x + y * y + z * z));
	}
	__device__
	float dot(const Vec &b) const {
		return x * b.x + y * b.y + z * b.z;
	} // cross:
	__device__
	Vec operator%(Vec &b) {
		return Vec(y * b.z - z * b.y, z * b.x - x * b.z, x * b.y - y * b.x);
	}
};

struct Ray {
	Vec o, d;
        __device__
	Ray(Vec o_, Vec d_) : o(o_), d(d_) {}
};

enum Refl_t { DIFF, SPEC, REFR }; // material types, used in radiance()

struct Sphere {
	float rad;   // radius
	Vec p, e, c; // position, emission, color
	Refl_t refl; // reflection type (DIFFuse, SPECular, REFRactive)
	__host__ __device__
	Sphere(float rad_, Vec p_, Vec e_, Vec c_, Refl_t refl_)
		: rad(rad_), p(p_), e(e_), c(c_), refl(refl_) {}
	__device__
	inline float intersect(const Ray &r) const { // returns distance, 0 if nohit
		Vec op = p - r.o; // Solve t^2*d.d + 2*t*(o-p).d + (o-p).(o-p)-R^2 = 0
		float t, eps = 1.5e-2f, b = op.dot(r.d),
		      det = b * b - op.dot(op) + rad * rad;
		if (det < 0)
			return 0;
		else
			det = sqrt(det);
		return (t = b - det) > eps ? t : ((t = b + det) > eps ? t : 0);
	}
};

__host__ __device__
inline float clamp(float x) { return x < 0 ? 0 : x > 1 ? 1 : x; }

inline int toInt(float x) {
	return int(pow(clamp(x), 1 / 2.2f) * 255 + .5f);
}

template<typename T>
__device__
inline bool intersect(const Ray &r, float &t, int &id,
		T spheres) {
	float d, inf = t = 1e20f;
	for (int i = 9; i--;)
		if ((d = spheres[i].intersect(r)) && d < t) {
			t = d;
			id = i;
		}
	return t < inf;
}

template<typename T>
__device__
Vec radiance(const Ray &r_, int depth_, T spheres, RNG &rng) {
	float t;
	int id = 0;
	Ray r = r_;
	int depth = depth_;
	Vec cl(0, 0, 0); // accumulated color
	Vec cf(1, 1, 1); // accumulated reflectance
	while (1) {
		if (!intersect(r, t, id, spheres))
			return cl;                     // if miss, return black
		const Sphere &obj = spheres[id]; // the hit object
		Vec x = r.o + r.d * t, n = (x - obj.p).norm(),
		    nl = n.dot(r.d) < 0 ? n : n * -1, f = obj.c;
		float p = f.x > f.y && f.x > f.z ? f.x : f.y > f.z ? f.y : f.z; // max refl
		cl = cl + cf.mult(obj.e);
		if (++depth > 5) {
			if (rng() < p) {
				f = f * (1 / p);
			} else {
				return cl;
			}
		} // R.R.
		cf = cf.mult(f);
		if (obj.refl == DIFF) { // Ideal DIFFUSE reflection
			float r1 = 2 * M_PI * rng(), r2 = rng(), r2s = sqrt(r2);
			Vec w = nl,
			    u = ((fabs(w.x) > .1 ? Vec(0, 1) : Vec(1)) % w).norm(),
			    v = w % u;
			Vec d = (u * cos(r1) * r2s + v * sin(r1) * r2s +
					w * sqrt(1 - r2)).norm();
			r = Ray(x, d);
			continue;
		} else if (obj.refl == SPEC) { // Ideal SPECULAR reflection
			r = Ray(x, r.d - n * 2 * n.dot(r.d));
			continue;
		}
		Ray reflRay(x, r.d - n * 2 * n.dot(r.d)); // Ideal dielectric REFRACTION
		bool into = n.dot(nl) > 0;                // Ray from outside going in?
		float nc = 1, nt = 1.5, nnt = into ? nc / nt : nt / nc, ddn = r.d.dot(nl),
		      cos2t;
		if ((cos2t = 1 - nnt * nnt * (1 - ddn * ddn)) <
				0) { // Total internal reflection
			r = reflRay;
			continue;
		}
		Vec tdir =
			(r.d * nnt -
			 n * ((into ? 1 : -1) * (ddn * nnt + sqrt(cos2t)))).norm();
		float a = nt - nc, b = nt + nc, R0 = a * a / (b * b),
		      c = 1 - (into ? -ddn : tdir.dot(n));
		float Re = R0 + (1 - R0) * c * c * c * c * c, Tr = 1 - Re,
		      P = .25 + .5 * Re, RP = Re / P, TP = Tr / (1 - P);
		if (rng() < P) {
			cf = cf * RP;
			r = reflRay;
		} else {
			cf = cf * TP;
			r = Ray(x, tdir);
		}
		continue;
	}
}

__global__ void raytrace(Vec* c, const Sphere* spheres_, const int w, const int h, const int samps) {
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;
	Vec r;
	const Sphere *spheres = &spheres_[0];
	Ray cam(Vec(50, 52, 295.6), Vec(0, -0.042612, -1).norm()); // cam pos, dir
	Vec cx = Vec(w * .5135 / h), cy = (cx % cam.d).norm() * .5135;
	RNG rng(1 + (y * w) + x); // initialise our own rng with rand() seed
	for (int sy = 0, i = (h - y - 1) * w + x; sy < 2; sy++) // 2x2 subpixel rows
		for (int sx = 0; sx < 2; sx++, r = Vec()) {           // 2x2 subpixel cols
			for (int s = 0; s < samps; s++) {
				float r1 = 2 * rng(), dx = r1 < 1 ? sqrt(r1) - 1
					: 1 - sqrt(2 - r1);
				float r2 = 2 * rng(), dy = r2 < 1 ? sqrt(r2) - 1
					: 1 - sqrt(2 - r2);
				Vec d = cx * (((sx + .5 + dx) / 2 + x) / w - .5) +
					cy * (((sy + .5 + dy) / 2 + y) / h - .5) + cam.d;
				r = r + radiance(Ray(cam.o + d * 140, d.norm()), 0, spheres, rng) *
					(1. / samps);
			} // Camera rays are pushed ^^^^^ forward to start in interior
			c[i] = c[i] + Vec(clamp(r.x), clamp(r.y), clamp(r.z)) * .25;
		}
};

int main(int argc, char *argv[]) {

	const Sphere spheres_glob[] = {
		// Scene: radius, position, emission, color, material
		Sphere(1e4, Vec(1e4 + 1, 40.8, 81.6), Vec(), Vec(.75, .25, .25),
				DIFF), // Left
		Sphere(1e4, Vec(-1e4 + 99, 40.8, 81.6), Vec(), Vec(.25, .25, .75),
				DIFF),                                                     // Rght
		Sphere(1e4, Vec(50, 40.8, 1e4), Vec(), Vec(.75, .75, .75), DIFF), // Back
		Sphere(1e4, Vec(50, 40.8, -1e4 + 170), Vec(), Vec(), DIFF),       // Frnt
		Sphere(1e4, Vec(50, 1e4, 81.6), Vec(), Vec(.75, .75, .75), DIFF), // Botm
		Sphere(1e4, Vec(50, -1e4 + 81.6, 81.6), Vec(), Vec(.75, .75, .75),
				DIFF),                                                      // Top
		Sphere(16.5, Vec(27, 16.5, 47), Vec(), Vec(1, 1, 1) * .999, SPEC), // Mirr
		Sphere(16.5, Vec(73, 16.5, 78), Vec(), Vec(1, 1, 1) * .999, REFR), // Glas
		Sphere(600, Vec(50, 681.6 - .27, 81.6), Vec(12, 12, 12), Vec(),
				DIFF) // Lite
	};

	int w = 1024, h = 768; 
	int samps = 40; // # samples
	Vec *c = (Vec*) malloc(sizeof(Vec) * w * h);

	Vec *color_buffer;
	Sphere *spheres_buffer;

	hipMalloc((void**)&color_buffer, sizeof(Vec) * w * h);
	hipMalloc((void**)&spheres_buffer, sizeof(Sphere) * 9);

	hipMemcpy(spheres_buffer, &spheres_glob[0], sizeof(Sphere) * 9, hipMemcpyHostToDevice);

	raytrace<<< dim3(w/8, h/8), dim3(8, 8) >>>(color_buffer, spheres_buffer, w, h, samps);

	hipMemcpy(c, color_buffer, sizeof(Vec) * w * h, hipMemcpyDeviceToHost);

	FILE *f = fopen("image.ppm", "w"); // Write image to PPM file.
	fprintf(f, "P3\n%d %d\n%d\n", w, h, 255);
	for (int i = 0; i < w * h; i++)
		fprintf(f, "%d %d %d ", toInt(c[i].x), toInt(c[i].y), toInt(c[i].z));

	free(c);
	hipFree(color_buffer);
	hipFree(spheres_buffer);
	return 0;
}
